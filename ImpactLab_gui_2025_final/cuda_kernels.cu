#include "hip/hip_runtime.h"
// cuda_kernels.cu
#include "hip/hip_runtime.h"
#include ""
#include "cuda_kernels.h"
#include <algorithm> // Para std::min

// Kernel CUDA para aplicar o filtro S�pia
__global__ void sepiaKernel(unsigned char* input, unsigned char* output, int width, int height, int stride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        long offset = (long)y * stride + (long)x * 4; // Assumindo 32bpp (4 bytes: B, G, R, A)

        int b = input[offset];
        int g = input[offset + 1];
        int r = input[offset + 2];
        int a = input[offset + 3];

        float tr = 0.393f * r + 0.769f * g + 0.189f * b;
        float tg = 0.349f * r + 0.686f * g + 0.168f * b;
        float tb = 0.272f * r + 0.534f * g + 0.131f * b;

        // O C�DIGO CORRIGIDO:
        output[offset] = static_cast<unsigned char>(fminf(255.0f, tb));
        output[offset + 1] = static_cast<unsigned char>(fminf(255.0f, tg));
        output[offset + 2] = static_cast<unsigned char>(fminf(255.0f, tr));
        output[offset + 3] = a;
    }
}

// Kernel CUDA para aplicar o filtro de Invers�o de Cores
__global__ void invertKernel(unsigned char* input, unsigned char* output, int width, int height, int stride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        long offset = (long)y * stride + (long)x * 4; // Assumindo 32bpp (4 bytes: B, G, R, A)

        output[offset] = 255 - input[offset];     // Inverte o Azul
        output[offset + 1] = 255 - input[offset + 1]; // Inverte o Verde
        output[offset + 2] = 255 - input[offset + 2]; // Inverte o Vermelho
        output[offset + 3] = input[offset + 3];      // Preserva o canal alfa
    }
}


// Fun��o Wrapper para S�pia
void applySepiaFilterCuda(unsigned char* h_input, unsigned char* h_output, int width, int height, int stride) {
    unsigned char* d_input, * d_output;
    size_t imageSize = (size_t)stride * height;

    // 1. Alocar mem�ria na GPU
    hipMalloc((void**)&d_input, imageSize);
    hipMalloc((void**)&d_output, imageSize);

    // 2. Copiar imagem da CPU (host) para a GPU (device)
    hipMemcpy(d_input, h_input, imageSize, hipMemcpyHostToDevice);

    // 3. Configurar a grade de threads para o kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 4. Lan�ar o kernel na GPU
    sepiaKernel << <numBlocks, threadsPerBlock >> > (d_input, d_output, width, height, stride);

    // Sincronizar para garantir que o kernel terminou antes de copiar de volta
    hipDeviceSynchronize();

    // 5. Copiar a imagem processada da GPU para a CPU
    hipMemcpy(h_output, d_output, imageSize, hipMemcpyDeviceToHost);

    // 6. Liberar mem�ria da GPU
    hipFree(d_input);
    hipFree(d_output);
}


// Fun��o Wrapper para Invers�o de Cores
void applyInvertFilterCuda(unsigned char* h_input, unsigned char* h_output, int width, int height, int stride) {
    unsigned char* d_input, * d_output;
    size_t imageSize = (size_t)stride * height;

    hipMalloc((void**)&d_input, imageSize);
    hipMalloc((void**)&d_output, imageSize);

    hipMemcpy(d_input, h_input, imageSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    invertKernel << <numBlocks, threadsPerBlock >> > (d_input, d_output, width, height, stride);

    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, imageSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}